#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>


#include <sys/time.h>

using namespace std;


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}


int main()
{
    srand(time(0));
    int M = 2000;              //矩阵A的行，矩阵C的行
    int N = 16384;               //矩阵A的列，矩阵B的行
    int K = 16384;               //矩阵B的列，矩阵C的列

    float *h_A = (float*)malloc(sizeof(float)*M*N);
    float *h_B = (float*)malloc(sizeof(float)*N*K);
    float *h_C = (float*)malloc(sizeof(float)*M*K);

    for (int i = 0; i < M*N; i++) {
        h_A[i] = i;
        // cout << h_A[i] << "  ";
        // if ((i + 1) % N == 0)
        //     cout << endl;        
    }
    //  cout << endl;

    for (int i = 0; i < N*K; i++) {
        h_B[i] =i;
        // cout << h_B[i] << "  ";
        // if ((i + 1) % K == 0)
        //     cout << endl;
    }
    cout << endl;

    double iStart, iElaps;

    float *d_A, *d_B, *d_C,*d_CT;
    hipMalloc((void**)&d_A, sizeof(float)*M*N);
    hipMalloc((void**)&d_B, sizeof(float)*N*K);
    hipMalloc((void**)&d_C, sizeof(float)*M*K);
    hipMemcpy(d_A, h_A, M*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N*K * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1;
    float beta = 0;

    //C=A*B
    hipblasHandle_t handle;
    
    hipblasCreate(&handle);
    
    // clock_t start = clock();//MNK Bt*At
    
    iStart = seconds();


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipblasSgemm(handle,
        HIPBLAS_OP_N,  
        HIPBLAS_OP_N,   
        K,                    //矩阵B的列数
        M,                    //矩阵A的行数
        N,                    //矩阵A的列数
        &alpha,           
        d_B,            
        K,                    
        d_A,         
        N,         
        &beta,          
        d_C,           
        K);
    
    CHECK(hipGetLastError()) ;

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);


    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    elapsed /= 1000.0f;

    iElaps = seconds() - iStart;

    // clock_t end = clock();
    // double sum_time = double(double(end - start)/CLOCKS_PER_SEC) * 1000;
    

    printf("time= %lf\n", elapsed);

    // cout<<"inference time: "<< sum_time <<endl; 
    float teps = (2 *(long) M * N * K) / elapsed;
    cout << "TEPS = " << teps << endl; 

    hipMemcpy(h_C, d_C, M*K * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 1; i++)
    {
        cout << h_C[i] << "  ";
        // if ((i+1)%K==0)
        //     cout << endl;
    }
    cout << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}