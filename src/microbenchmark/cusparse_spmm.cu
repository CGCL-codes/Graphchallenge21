#include <hip/hip_runtime.h>
#include "../gpu_lib/header.h"
#include "../utils/header.h"
#include <stdio.h>
#include <algorithm>
#include <cstdlib>
#include <cstring>
#include <stdio.h>
#include <iostream>
#include <hipsparse.h>
#include <vector>

namespace ftxj {

#define CUSPARSE_CHECK(x) {hipsparseStatus_t _c=x; if (_c != HIPSPARSE_STATUS_SUCCESS) {printf("cusparse fail: %d, line: %d\n", (int)_c, __LINE__); exit(-1);}}


void test_benchmark_cusparse(COOMatrix& coo, cuSPARSEMatrix &matrix, int neuron, int batch) {

	float * input = (float*)malloc(sizeof(float) * neuron * batch);
	memset(input, 0, sizeof(float) * neuron * batch);

	float * output = (float*)malloc(sizeof(float) * neuron * batch);
	memset(output, 0, sizeof(float) * neuron * batch);


    srand (static_cast <unsigned> (time(0)));
    for(int i = 0; i < batch; ++i) {
      for(int j = 0; j < neuron; ++j) {
        float r2 = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/32.0));
        input[i * neuron + j] = r2;
      }
    }

    float *A_d;
    float *B_d;
    
    int* len_d;
    int* index_d;
    float* val_d;

    Safe_Call(hipMalloc((void**)&A_d, sizeof(float) * neuron * batch));
    Safe_Call(hipMemcpy(A_d, input, sizeof(float) * neuron * batch, hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&B_d, sizeof(float) * neuron * batch));
    Safe_Call(hipMemset(B_d, 0, sizeof(float) * neuron * batch));


    Safe_Call(hipMalloc((void**)&len_d, sizeof(int) * (neuron + 1)));
    Safe_Call(hipMemcpy(len_d, matrix.len, sizeof(int) * (neuron + 1), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&index_d, sizeof(int) * (neuron * 32)));
    Safe_Call(hipMemcpy(index_d, matrix.index, sizeof(int) * (neuron * 32), hipMemcpyHostToDevice));
  
    Safe_Call(hipMalloc((void**)&val_d, sizeof(float) * (neuron * 32)));
    Safe_Call(hipMemcpy(val_d, matrix.val, sizeof(float) * (neuron * 32), hipMemcpyHostToDevice));



    hipsparseHandle_t handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void* dBuffer    = NULL;
    size_t bufferSize = 0;
    float alpha = 1.0f;
    float beta = 0.0f;

    CUSPARSE_CHECK( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format

    CUSPARSE_CHECK(hipsparseCreateCsr(&matA, neuron, neuron, 32 * neuron,
                                      len_d, index_d, val_d,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    // Create dense matrix B
    CUSPARSE_CHECK(hipsparseCreateDnMat(&matB, neuron, batch, neuron, A_d,
                                    HIP_R_32F, HIPSPARSE_ORDER_COL) )
                                        
    // Create dense matrix C
    CUSPARSE_CHECK(hipsparseCreateDnMat(&matC, neuron, batch, neuron, B_d,
                                    HIP_R_32F, HIPSPARSE_ORDER_COL) )
    

                                        
    Safe_Call(hipMalloc(&dBuffer, bufferSize));
              

    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
                                handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
   
                                HIPSPARSE_CSRMM_ALG1, &bufferSize) )
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

   
    CUSPARSE_CHECK( hipsparseSpMM(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_MM_ALG_DEFAULT, dBuffer) )
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop); //ms 

    // destroy matrix/vector descriptors
    CUSPARSE_CHECK( hipsparseDestroySpMat(matA) )
    CUSPARSE_CHECK( hipsparseDestroyDnMat(matB) )
    CUSPARSE_CHECK( hipsparseDestroyDnMat(matC) )
    CUSPARSE_CHECK( hipsparseDestroy(handle) )

    Safe_Call(hipMemcpy(output, B_d, neuron * batch  * sizeof(float), hipMemcpyDeviceToHost));

	std::cout << "kernel time = " << elapsed << "ms" << std::endl;
    std::cout << "Flops [cuSparse] = " << float(2 * batch * neuron * 32) /  elapsed * 1000 /1e12 << "TFLOPS"<< std::endl;

	CpuSpmm::run_and_cmp(coo, input, neuron, batch, output, false);

}
}
