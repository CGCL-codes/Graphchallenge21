#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../gpu_lib/header.h"
#include "../utils/header.h"
#include <cstdio>
#include <cstdlib>
namespace ftxj {

__device__ inline float __ReLU(float x) {
   return x<0.0?0.0:x>32.0?32.0:x;
};

__global__ void batch_parallel_16384x32succ_kernel(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C, int* __restrict__ index, int neuron, int batch, float bias) {
	extern __shared__ float shared[];
	float reduce[32] = {0.0};

	for(int n = threadIdx.x; n < 32 * 32; n += blockDim.x){
		shared[n] = B[(blockIdx.y * 32 * 32) + n];
	}
	__syncthreads();
	if((blockIdx.x * blockDim.x + threadIdx.x) >= batch) return;
	
	for(int r = 0; r < 32; ++r) {
		int row_idx = index[blockIdx.y * 32 + r];
		float val = A[row_idx * batch + blockIdx.x * blockDim.x + threadIdx.x];
		for(int c = 0; c < 32; ++c){
			reduce[c] += shared[r * 32 + c] * val;
		}
	}
	__syncthreads();
	for(int c = 0; c < 16; ++c) {
		C[(blockIdx.y * 16  + c) * batch + blockIdx.x * blockDim.x + threadIdx.x] = reduce[c];
	}
	for(int c = 16; c < 32; ++c) {
		C[(neuron / 2 + blockIdx.y * 16  + c - 16) * batch + blockIdx.x * blockDim.x + threadIdx.x] = reduce[c];
	}
}

void test_benchmark_row_succ_input_transpose_batch_parallel(COOMatrix& coo, std::vector<float> &val, std::vector<int> &row_access, int batch, int neuron, GpuEnv &env) {

	float *A;
    float *B;
	float *C;
	int *index;

	int mybatch = batch;

	int bias = 0;

	float * input = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(input, 0, sizeof(float) * neuron * mybatch);

	float * output = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(output, 0, sizeof(float) * neuron * mybatch);

	srand (static_cast <unsigned> (time(0)));
	for(int i = 0; i < mybatch; ++i) {
		for(int j = 0; j < neuron; ++j) {
            float r2 = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/32.0));
			input[i * neuron + j] = r2;
		}
	}


	float* W  = (float*)malloc(sizeof(float) * val.size());
	for(int i = 0; i < val.size(); ++i) {
		W[i] = val[i];
	}

	int* access = (int*)malloc(sizeof(int) * row_access.size());
	for(int i = 0; i < row_access.size(); ++i) {
		access[i] = row_access[i];
	}


    Safe_Call(hipMalloc((void**)&A, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemcpy(A, input, sizeof(float) * neuron * mybatch, hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&B, sizeof(float) * val.size()));
    Safe_Call(hipMemcpy(B, W, sizeof(float) * val.size(), hipMemcpyHostToDevice));

	Safe_Call(hipMalloc((void**)&C, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemset(C, 0, sizeof(float) * neuron * mybatch));

	Safe_Call(hipMalloc((void**)&index, sizeof(int) * row_access.size()));
	Safe_Call(hipMemcpy(index, access, sizeof(int) * row_access.size(), hipMemcpyHostToDevice));

	env.add_event("row-succ-20-uiuc-kernel");
    env.event_start_record("row-succ-20-uiuc-kernel");

	int blocksize = 256;
	dim3 block(blocksize);
    dim3 grid((mybatch + blocksize - 1) / blocksize,  neuron / 32);

	batch_parallel_16384x32succ_kernel<<<grid, block, sizeof(float) * (32 * 32), env.get_stream("row-succ-20-uiuc-kernel")>>>(
		A, B, C, index, neuron, batch, bias
	);

    env.event_stop_record("row-succ-20-uiuc-kernel");

    float time = env.get_event_time("row-succ-20-uiuc-kernel"); 

	Safe_Call(hipMemcpy(output, C, sizeof(float) * neuron * mybatch, hipMemcpyDeviceToHost));

	std::cout << "Kernel Exec Time [20-uiuc-row-succ-transpose] = " << time <<  "ms" <<std::endl;
	std::cout << "Kernel Exec Flops = " << (neuron * mybatch * 32 * 2.0) / (time / 1000.0) / 1000 / 1000 / 1000 /1000 << "TFLOPS" <<std::endl;

	CpuSpmm::run_and_cmp(coo, input, neuron, mybatch, output, false, false, false);

	
}
};