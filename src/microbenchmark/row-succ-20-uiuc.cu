#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../gpu_lib/header.h"
#include "../utils/header.h"
#include <cstdio>
#include <cstdlib>
namespace ftxj {

__device__ inline float __ReLU(float x){
   return x<0.0?0.0:x>32.0?32.0:x;
};

#define MINIBATCH 32

__global__ void uiuc_cut_kernel(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C, int* __restrict__ index, int neuron, int batch, float bias) {

	extern __shared__ float shared[];
	float reduce[MINIBATCH] = {0.0};


	int idx = index[(blockIdx.y * blockDim.x + threadIdx.x) / 32 + threadIdx.x % 32];

	for(unsigned int f = 0; f < MINIBATCH; f++) {
		shared[f * blockDim.x + threadIdx.x] = A[(blockIdx.x * MINIBATCH + f) * neuron + idx];
	}
	__syncthreads();
	for(int r = 0; r < 32; ++r){
		float val = B[blockIdx.y * blockDim.x * 32 + r * blockDim.x + threadIdx.x];
		for(int f = 0; f < MINIBATCH; f++) {
			// if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 1 && f == 0) {
			// 	printf("%f * %f\n", shared[f * blockDim.x + (threadIdx.x / 32+ r)], val);
			// }
			reduce[f] += shared[f * blockDim.x + (threadIdx.x / 32 + r)] * val;
		}
	}

	// if(blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 1) {
	// 	printf("res = %f\n", reduce[0]);
	// }

	int widx1 = (blockIdx.y * blockDim.x) / 2;
	int widx2 = (blockIdx.y * blockDim.x) / 2 + 512;
	int wgroup = threadIdx.x / 32;

	int widx = threadIdx.x % 32 > 16 ? widx2 + wgroup * 16 + threadIdx.x % 32 - 16 : widx1 + wgroup * 16 + threadIdx.x % 32;

	for(int f = 0; f < MINIBATCH; f++) {
		C[(blockIdx.x * MINIBATCH + f) * neuron + widx] = reduce[f];
	}
}

void test_benchmark_row_succ_20_uiuc(COOMatrix& coo, std::vector<float> &val, std::vector<int> &row_access, int batch, int neuron, GpuEnv &env) {

	float *A;
    float *B;
	float *C;
	int *index;

	int mybatch = batch;

	int bias = 0;

	float * input = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(input, 0, sizeof(float) * neuron * mybatch);

	float * output = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(output, 0, sizeof(float) * neuron * mybatch);

	srand (static_cast <unsigned> (time(0)));
	for(int i = 0; i < mybatch; ++i) {
		for(int j = 0; j < neuron; ++j) {
            float r2 = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/32.0));
			input[i * neuron + j] = r2;
		}
	}


	float* W  = (float*)malloc(sizeof(float) * val.size());
	for(int i = 0; i < val.size(); ++i) {
		W[i] = val[i];
	}

	int* access = (int*)malloc(sizeof(int) * row_access.size());
	for(int i = 0; i < row_access.size(); ++i) {
		access[i] = row_access[i];
	}


    Safe_Call(hipMalloc((void**)&A, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemcpy(A, input, sizeof(float) * neuron * mybatch, hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&B, sizeof(float) * val.size()));
    Safe_Call(hipMemcpy(B, W, sizeof(float) * val.size(), hipMemcpyHostToDevice));

	Safe_Call(hipMalloc((void**)&C, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemset(C, 0, sizeof(float) * neuron * mybatch));

	Safe_Call(hipMalloc((void**)&index, sizeof(int) * row_access.size()));
	Safe_Call(hipMemcpy(index, access, sizeof(int) * row_access.size(), hipMemcpyHostToDevice));

	env.add_event("row-succ-20-uiuc-kernel");
    env.event_start_record("row-succ-20-uiuc-kernel");

	int blocksize = 256;
	dim3 block(blocksize);
    dim3 grid(mybatch / (MINIBATCH), neuron / blocksize);

	uiuc_cut_kernel<<<grid, block, sizeof(float) * (MINIBATCH * blocksize), env.get_stream("row-succ-20-uiuc-kernel")>>>(
		A, B, C, index, neuron, batch, bias
	);

    env.event_stop_record("row-succ-20-uiuc-kernel");

    float time = env.get_event_time("row-succ-20-uiuc-kernel"); 

	Safe_Call(hipMemcpy(output, C, sizeof(float) * neuron * mybatch, hipMemcpyDeviceToHost));

	std::cout << "Kernel Exec Time [20-uiuc-row-succ] = " << time <<  "ms" <<std::endl;
	std::cout << "Kernel Exec Flops = " << (neuron * mybatch * 32 * 2.0) / (time / 1000.0) / 1000 / 1000 / 1000 /1000 << "TFLOPS" <<std::endl;

	CpuSpmm::run_and_cmp(coo, input, neuron, mybatch, output, false, true);

	
}
};