#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../gpu_lib/header.h"
#include "../utils/header.h"
#include <stdio.h>
#include <algorithm>
#include <cstdlib>
#include <cstring>

namespace ftxj {
__device__ float __ReLU(float x){
   return x<0.0?0.0:x>32.0?32.0:x;
};

#define WARPSIZE 32
#define MINIBATCH 12

__global__ void __launch_bounds__(1024,1) dummy_kernel(
  float *nextfeat, float *currfeat, 
  int buffsize, int *buffdispl, int *mapdispl, unsigned short *map, 
  int *displ, unsigned short *index, float *value, 
  float bias, int neuron
){
	extern __shared__ float shared[];
	int wind = threadIdx.x % WARPSIZE;
	float reduce[MINIBATCH] = {0.0};
	for(int buff = buffdispl[blockIdx.x]; buff < buffdispl[blockIdx.x+1]; buff++){
		int mapnz = mapdispl[buff+1]-mapdispl[buff];
		for(int n = threadIdx.x; n < mapnz; n += blockDim.x){
			int ind = map[mapdispl[buff]+n];
			for(unsigned int f = 0; f < MINIBATCH; f++) {
				shared[f*buffsize+n] = currfeat[(blockIdx.y * MINIBATCH+f) * (unsigned int) neuron+ind];
			}
    	}
		__syncthreads();
		int warp = (buff*blockDim.x+threadIdx.x)/WARPSIZE;
		for(int m = displ[warp]; m < displ[warp+1]; m++){
			int ind = index[m*WARPSIZE+wind];
			float val = value[m*WARPSIZE+wind];
			for(int f = 0; f < MINIBATCH; f++) {
				reduce[f] += shared[f*buffsize+ind] * val;
			}
		}
		__syncthreads();
	}
	int m = blockIdx.x*blockDim.x+threadIdx.x;

	for(int f = 0; f < MINIBATCH; f++)
		nextfeat[(blockIdx.y * MINIBATCH + f) * neuron + m] = __ReLU(reduce[f]+bias);
    
};

void uiuc_test_benchmark(COOMatrix &coo, UIUCMatrix &matrix, GpuEnv &env) {
    float *nextfeat;
    float *currfeat;

    int buffsize = matrix.buffsize;
    int neuron = matrix.neuron;

    int *buffdispl; 
    int *mapdispl;
    unsigned short *map; 
    int *displ;
    unsigned short *index;
    float *value; 
    float bias = 0;

    int mybatch = 1800;

    // std::vector<std::vector<float>> input(mybatch, std::vector<float>(neuron, 0.0));

	float * input = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(input, 0, sizeof(float) * neuron * mybatch);

	float * output = (float*)malloc(sizeof(float) * neuron * mybatch);
	memset(output, 0, sizeof(float) * neuron * mybatch);


	for(int i = 0; i < mybatch; ++i) {
		for(int j = 0; j < neuron; ++j) {
			input[i * neuron + j] = 1.0;
		}
	}
  
    Safe_Call(hipMalloc((void**)&buffdispl, sizeof(int) * matrix.buffdispl.size()));
    Safe_Call(hipMemcpy(buffdispl, &matrix.buffdispl[0], sizeof(int) * matrix.buffdispl.size(), hipMemcpyHostToDevice));
    
    Safe_Call(hipMalloc((void**)&mapdispl, sizeof(int) * matrix.mapdispl.size()));
    Safe_Call(hipMemcpy(mapdispl, &matrix.mapdispl[0], sizeof(int) * matrix.mapdispl.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&map, sizeof(unsigned short) * matrix.map.size()));
    Safe_Call(hipMemcpy(map, &matrix.map[0], sizeof(unsigned short) * matrix.map.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&displ, sizeof(int) * matrix.warpdispl.size()));
    Safe_Call(hipMemcpy(displ, &matrix.warpdispl[0], sizeof(int) * matrix.warpdispl.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&index, sizeof(unsigned short) * matrix.warpindex.size()));
    Safe_Call(hipMemcpy(index, &matrix.warpindex[0], sizeof(unsigned short) * matrix.warpindex.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&value, sizeof(float) * matrix.warpvalue.size()));
    Safe_Call(hipMemcpy(value, &matrix.warpvalue[0], sizeof(float) * matrix.warpvalue.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&currfeat, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemcpy(currfeat, input, sizeof(float) * neuron * mybatch, hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&nextfeat, sizeof(float) * neuron * mybatch));
    Safe_Call(hipMemset(nextfeat, 0, sizeof(float) * neuron * mybatch));

    env.add_event("kernel_timer");
    env.event_start_record("kernel_timer");

    dim3 block(matrix.blocksize);
    dim3 grid(neuron / matrix.blocksize, (mybatch+MINIBATCH-1)/MINIBATCH);
    dummy_kernel<<<grid,block, sizeof(float) * matrix.buffsize * MINIBATCH, env.get_stream("kernel_timer")>>>(
        nextfeat, currfeat, buffsize, buffdispl, mapdispl, map, displ, index, value,
        bias, neuron
    );

    env.event_stop_record("kernel_timer");
    float time = env.get_event_time("kernel_timer"); 

    Safe_Call(hipMemcpy(output, nextfeat, sizeof(float) * neuron * mybatch, hipMemcpyDeviceToHost));
    
	CpuSpmm::run_and_cmp(coo, input, neuron, mybatch, output);

    std::cout << "uiuc timer = " << time << std::endl;
}
};