#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../gpu_lib/header.h"
#include "../utils/header.h"
namespace ftxj {
__device__ float __ReLU(float x){
   return x<0.0?0.0:x>32.0?32.0:x;
};

#define WARPSIZE 32
#define MINIBATCH 12

__global__ void __launch_bounds__(1024,1) dummy_kernel(
  float *nextfeat, float *currfeat, 
  int buffsize, int *buffdispl, int *mapdispl, unsigned short *map, 
  int *displ, unsigned short *index, float *value, 
  float bias, int neuron
){
  extern __shared__ float shared[];
  int wind = threadIdx.x % WARPSIZE;
  float reduce[MINIBATCH] = {0.0};
  for(int buff = buffdispl[blockIdx.x]; buff < buffdispl[blockIdx.x+1]; buff++){
    int mapnz = mapdispl[buff+1]-mapdispl[buff];
    for(int n = threadIdx.x; n < mapnz; n += blockDim.x){
      int ind = map[mapdispl[buff]+n];
      for(unsigned int f = 0; f < MINIBATCH; f++)
        shared[f*buffsize+n] = currfeat[(blockIdx.y * MINIBATCH+f) * (unsigned int) neuron+ind];
    }
    __syncthreads();
    int warp = (buff*blockDim.x+threadIdx.x)/WARPSIZE;
    for(int m = displ[warp]; m < displ[warp+1]; m++){
      int ind = index[m*WARPSIZE+wind];
      float val = value[m*WARPSIZE+wind];
      for(int f = 0; f < MINIBATCH; f++)
        reduce[f] += shared[f*buffsize+ind]*val;
    }
    __syncthreads();
  }
  int m = blockIdx.x*blockDim.x+threadIdx.x;
  for(int f = 0; f < MINIBATCH; f++)
    nextfeat[(blockIdx.y*MINIBATCH+f) * neuron + m] = __ReLU(reduce[f]+bias);
    
};

void uiuc_test_benchmark(UIUCMatrix &matrix, GpuEnv &env) {
    float *nextfeat;
    float *currfeat;

    int buffsize = matrix.buffsize;
    int neuron = matrix.neuron;

    int *buffdispl; 
    int *mapdispl;
    unsigned short *map; 
    int *displ;
    unsigned short *index;
    float *value; 
    float bias = -0.3;

    int mybatch = 1800;

    std::vector<std::vector<float>> input(mybatch, std::vector<float>(neuron, 1.0));

    Safe_Call(hipMalloc((void**)&buffdispl, sizeof(int) * matrix.buffdispl.size()));
    Safe_Call(hipMemcpy(buffdispl, &matrix.buffdispl[0], sizeof(int) * matrix.buffdispl.size(), hipMemcpyHostToDevice));
    
    Safe_Call(hipMalloc((void**)&mapdispl, sizeof(int) * matrix.mapdispl.size()));
    Safe_Call(hipMemcpy(mapdispl, &matrix.mapdispl[0], sizeof(int) * matrix.mapdispl.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&map, sizeof(unsigned short) * matrix.map.size()));
    Safe_Call(hipMemcpy(map, &matrix.map[0], sizeof(unsigned short) * matrix.map.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&displ, sizeof(int) * matrix.warpdispl.size()));
    Safe_Call(hipMemcpy(displ, &matrix.warpdispl[0], sizeof(int) * matrix.warpdispl.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&index, sizeof(unsigned short) * matrix.warpindex.size()));
    Safe_Call(hipMemcpy(index, &matrix.warpindex[0], sizeof(unsigned short) * matrix.warpindex.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&value, sizeof(float) * matrix.warpvalue.size()));
    Safe_Call(hipMemcpy(value, &matrix.warpvalue[0], sizeof(float) * matrix.warpvalue.size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&currfeat, sizeof(float) * input.size() * input[0].size()));
    Safe_Call(hipMemcpy(currfeat, &input[0][0], sizeof(float) * input.size() * input[0].size(), hipMemcpyHostToDevice));

    Safe_Call(hipMalloc((void**)&nextfeat, sizeof(float) * input.size() * input[0].size()));
    Safe_Call(hipMemset(nextfeat, 0, sizeof(float) * input.size() * input[0].size()));


    

    env.add_event("kernel_timer");
    env.event_start_record("kernel_timer");

    dim3 block(matrix.blocksize);
    dim3 grid(neuron / matrix.blocksize, (mybatch+MINIBATCH-1)/MINIBATCH);
    dummy_kernel<<<grid,block, sizeof(float) * matrix.buffsize * MINIBATCH, env.get_stream("kernel_timer")>>>(
        nextfeat, currfeat, buffsize, buffdispl, mapdispl, map, displ, index, value,
        bias, neuron
    );

    env.event_stop_record("kernel_timer");
    float time = env.get_event_time("kernel_timer"); 
    std::cout << "uiuc timer = " << time << std::endl;
}
};