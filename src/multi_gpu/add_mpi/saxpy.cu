#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add_kernel(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void handle(int gpu_number)
{
  int N_GPU;
  hipGetDeviceCount(&N_GPU);
  //printf("gpu count : %d\n",N_GPU);
 
   //Arrange the task of each GPU
  int N = ((1<<30)+N_GPU - 1)/N_GPU;

  hipSetDevice(gpu_number);

  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  float time_elapsed=0;
  hipEvent_t start,stop;
  hipEventCreate(&start);    //创建Event
  hipEventCreate(&stop);
  hipEventRecord( start,0);    //记录当前时间

  // Perform SAXPY on 1M elements
  add_kernel<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop,0);    //记录当前时间
  hipEventSynchronize(start);    //Waits for an event to complete.
  hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);
  hipEventDestroy(start);    //destory the event
  hipEventDestroy(stop);
  printf("card%d 执行时间：%f(ms)\n",gpu_number,time_elapsed);
}